#include "hip/hip_runtime.h"
#include "./gaussian_kernel.h" 

/*
The actual gaussian blur kernel to be implemented by 
you. Keep in mind that the kernel operates on a 
single channel.
 */
__global__ 
void gaussianBlur(unsigned char *d_in, unsigned char *d_out, 
        const int rows, const int cols, float *d_filter, const int filterWidth){


} 



/*
  Given an input RGBA image separate 
  that into appropriate rgba channels.
 */
__global__ 
void separateChannels(uchar4 *d_imrgba, unsigned char *d_r, unsigned char *d_g, unsigned char *d_b,
        const int rows, const int cols){


} 
 

/*
  Given input channels combine them 
  into a single uchar4 channel. 

  You can use some handy constructors provided by the 
  cuda library i.e. 
  make_int2(x, y) -> creates a vector of type int2 having x,y components 
  make_uchar4(x,y,z,255) -> creates a vector of uchar4 type x,y,z components 
  the last argument being the transperency value. 
 */
__global__ 
void recombineChannels(unsigned char *d_r, unsigned char *d_g, unsigned char *d_b, uchar4 *d_orgba,
        const int rows, const int cols){


} 


void your_gauss_blur(uchar4* d_imrgba, uchar4 *d_oimrgba, size_t rows, size_t cols, 
        unsigned char *d_red, unsigned char *d_green, unsigned char *d_blue, 
        unsigned char *d_rblurred, unsigned char *d_gblurred, unsigned char *d_bblurred,
        float *d_filter,  int filterWidth){
 


        dim3 blockSize(1,1,1);
        dim3 gridSize(1,1,1);

        separateChannels<<<gridSize, blockSize>>>(d_imrgba, d_red, d_green, d_blue, rows, cols);
        hipDeviceSynchronize();
        checkCudaErrors(hipGetLastError());

        gaussianBlur<<<gridSize, blockSize>>>(d_red, d_rblurred, rows, cols, d_filter, filterWidth);
        hipDeviceSynchronize();
        checkCudaErrors(hipGetLastError());

        gaussianBlur<<<gridSize, blockSize>>>(d_green, d_gblurred, rows, cols, d_filter, filterWidth);
        hipDeviceSynchronize();
        checkCudaErrors(hipGetLastError());

        gaussianBlur<<<gridSize, blockSize>>>(d_blue, d_bblurred, rows, cols, d_filter, filterWidth);
        hipDeviceSynchronize();
        checkCudaErrors(hipGetLastError());

        recombineChannels<<<gridSize, blockSize>>>(d_rblurred, d_gblurred, d_bblurred, d_oimrgba, rows, cols);

        hipDeviceSynchronize();
        checkCudaErrors(hipGetLastError());   

}




